#include "hip/hip_runtime.h"
#include "common.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <nccl.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

void nccl_check(ncclResult_t status, const char *file, int line) {
  if (status != ncclSuccess) {
    printf("[NCCL ERROR] at file %s:%d:\n%s\n", file, line,
           ncclGetErrorString(status));
    exit(EXIT_FAILURE);
  }
}
#define ncclCheck(err) (nccl_check(err, __FILE__, __LINE__))

void mpi_check(int status, const char *file, int line) {
  if (status != MPI_SUCCESS) {
    char mpi_error[4096];
    int mpi_error_len = 0;
    assert(MPI_Error_string(status, &mpi_error[0], &mpi_error_len) ==
           MPI_SUCCESS);
    printf("[MPI ERROR] at file %s:%d:\n%.*s\n", file, line, mpi_error_len,
           mpi_error);
    exit(EXIT_FAILURE);
  }
}
#define mpiCheck(err) (mpi_check(err, __FILE__, __LINE__))

// Sets a vector to a predefined value
__global__ void set_vector(float *data, int N, float value) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // Check for out-of-bounds access
  if (i < N) {
    data[i] = value;
  }
}

size_t cdiv(size_t a, size_t b) { return (a + b - 1) / b; }

struct NCCL {
  int process_rank;
  int mpi_world_size;
  ncclComm_t nccl_comm;
  hipStream_t stream;
};

void ncclInit(int *argc, char ***argv, NCCL *nccl) {
  // Initialize MPI.
  mpiCheck(MPI_Init(argc, argv));
  mpiCheck(MPI_Comm_rank(MPI_COMM_WORLD, &nccl->process_rank));
  mpiCheck(MPI_Comm_size(MPI_COMM_WORLD, &nccl->mpi_world_size));

  // This process manages the defined device.
  cudaCheck(hipSetDevice(nccl->process_rank));

  printf("Current process rank is: %d/%d\n", nccl->process_rank,
         nccl->mpi_world_size);

  // Generate and broadcast a unique NCCL ID for initialization.
  ncclUniqueId nccl_id;
  if (nccl->process_rank == 0) {
    ncclCheck(ncclGetUniqueId(&nccl_id));
  }
  mpiCheck(MPI_Bcast((void *)&nccl_id, sizeof(nccl_id), MPI_BYTE, 0,
                     MPI_COMM_WORLD));

  ncclCheck(ncclCommInitRank(&nccl->nccl_comm, nccl->mpi_world_size, nccl_id,
                             nccl->process_rank));

  // Create a stream for cross-device operations.
  cudaCheck(hipStreamCreate(&nccl->stream));
}

void ncclDestroy(NCCL *nccl) {
  cudaCheck(hipStreamDestroy(nccl->stream));
  ncclCommDestroy(nccl->nccl_comm);
  mpiCheck(MPI_Finalize());
}

float get_mean(float *arr, size_t size, int process_rank) {
  double sum = 0.0;
  for (size_t i = 0; i < size; ++i) {
    sum += arr[i];
  }
  return sum / size;
}

int main(int argc, char **argv) {
  // Some constants
  const size_t all_reduce_buffer_size = 32 * 1024 * 1024;
  const size_t threads_per_block = 1024;

  NCCL nccl;
  ncclInit(&argc, &argv, &nccl);

  // Allocating buffers on each of the devices.
  float *all_reduce_buffer;
  cudaCheck(
      hipMalloc(&all_reduce_buffer, all_reduce_buffer_size * sizeof(float)));

  int n_blocks = cdiv(all_reduce_buffer_size, threads_per_block);
  // Set the allocated memory to a defined value.
  set_vector<<<n_blocks, threads_per_block, 0, nccl.stream>>>(
      all_reduce_buffer, all_reduce_buffer_size,
      (float)(nccl.process_rank + 1));
  cudaCheck(hipGetLastError());

  float *all_reduce_buffer_host =
      (float *)malloc(all_reduce_buffer_size * sizeof(float));

  cudaCheck(hipMemcpy(all_reduce_buffer_host, all_reduce_buffer,
                       sizeof(float) * all_reduce_buffer_size,
                       hipMemcpyDeviceToHost));

  printf("Process rank %d: average value is %.6f\n", nccl.process_rank,
         get_mean(all_reduce_buffer_host, all_reduce_buffer_size,
                  nccl.process_rank));

  float *all_reduce_buffer_recv;
  cudaCheck(hipMalloc(&all_reduce_buffer_recv,
                       all_reduce_buffer_size * sizeof(float)));

  ncclCheck(ncclAllReduce(
      (const void *)all_reduce_buffer, (void *)all_reduce_buffer_recv,
      all_reduce_buffer_size, ncclFloat, ncclSum, nccl.nccl_comm, nccl.stream));

  hipStreamSynchronize(nccl.stream);

  cudaCheck(hipMemcpy(all_reduce_buffer_host, all_reduce_buffer_recv,
                       sizeof(float) * all_reduce_buffer_size,
                       hipMemcpyDeviceToHost));

  printf("Process rank %d: average value is %.6f\n", nccl.process_rank,
         get_mean(all_reduce_buffer_host, all_reduce_buffer_size,
                  nccl.process_rank));

  free(all_reduce_buffer_host);
  cudaCheck(hipFree(all_reduce_buffer));
  ncclDestroy(&nccl);
}

/*
#define MPICHECK(cmd)                                                          \
  do {                                                                         \
    int e = cmd;                                                               \
    if (e != MPI_SUCCESS) {                                                    \
      printf("Failed: MPI error %s:%d '%d'\n", __FILE__, __LINE__, e);         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define CUDACHECK(cmd)                                                         \
  do {                                                                         \
    hipError_t e = cmd;                                                       \
    if (e != hipSuccess) {                                                    \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__,            \
             hipGetErrorString(e));                                           \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define NCCLCHECK(cmd)                                                         \
  do {                                                                         \
    ncclResult_t r = cmd;                                                      \
    if (r != ncclSuccess) {                                                    \
      printf("Failed, NCCL error %s:%d '%s'\n", __FILE__, __LINE__,            \
             ncclGetErrorString(r));                                           \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

static uint64_t getHostHash(const char *string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++) {
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}

static void getHostName(char *hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i = 0; i < maxlen; i++) {
    if (hostname[i] == '.') {
      hostname[i] = '\0';
      return;
    }
  }
}

int main(int argc, char *argv[]) {
  int size = 32 * 1024 * 1024;

  int myRank, nRanks, localRank = 0;

  // initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  // calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
                         sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p = 0; p < nRanks; p++) {
    if (p == myRank)
      break;
    if (hostHashs[p] == hostHashs[myRank])
      localRank++;
  }

  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;

  // get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0)
    ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  // picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipStreamCreate(&s));

  // initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));

  const size_t threads_per_block = 1024;
  int n_blocks = cdiv(size, threads_per_block);
  // Set the allocated memory to a defined value.
  set_vector<<<n_blocks, threads_per_block, 0, s>>>(sendbuff, size,
                                                    (float)(myRank + 1));

  // communicating using NCCL
  NCCLCHECK(ncclAllReduce((const void *)sendbuff, (void *)recvbuff, size,
                          ncclFloat, ncclSum, comm, s));

  // completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  float *all_reduce_buffer_host = (float *)malloc(size * sizeof(float));

  cudaCheck(hipMemcpy(all_reduce_buffer_host, recvbuff, sizeof(float) * size,
                       hipMemcpyDeviceToHost));

  printf("Process rank %d: average value is %.6f\n", myRank,
         get_mean(all_reduce_buffer_host, size, myRank));

  // free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));

  // finalizing NCCL
  ncclCommDestroy(comm);

  // finalizing MPI
  MPICHECK(MPI_Finalize());

  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}
*/